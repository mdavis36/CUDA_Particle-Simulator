#include "hip/hip_runtime.h"
#include "ParticleSystem.h"


extern "C" {
      #include <stdio.h>
      #define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
      inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
      {
         if (code != hipSuccess)
         {
            fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
            if (abort) exit(code);
         }
      }
}

__global__ void cpyParticleDataToDraw( Particle* particles, vec3* positions, int num_particles )
{
      int indx = threadIdx.x + blockIdx.x * blockDim.x;
      if (indx >= num_particles) return;
      positions[indx] = particles[indx].x;
}

__global__ void initParticles( Particle* particles, int num_particles )
{
      int indx = threadIdx.x + blockIdx.x * blockDim.x;
      if (indx >= num_particles) return;

      int x_limit = 5;
      int z_limit = 5;

      hiprandState state;
      hiprand_init( (unsigned long long)clock() + indx, 0, 0, &state );

      vec3 r;
      r.x = -x_limit + (hiprand_uniform(&state) * x_limit * 2);
      r.z = -z_limit + (hiprand_uniform(&state) * z_limit * 2);
      r.y = 5 + (hiprand_uniform(&state) * 8);

      particles[indx] = Particle( 1, r, vec3(0.0f,0.0f,0.0f), vec3(0.0f,0.0f,0.0f) );
}

ParticleSystem::ParticleSystem()
{
      *this = ParticleSystem(1, PARTICLE_CUBE);
}

ParticleSystem::ParticleSystem(int n, int form)
{
      t = 0.0f;
      _num_particles = n;


      // ---------- Initialize CUDA Data ----------
      block = dim3(512);
      grid = dim3(std::ceil(n / block.x));
      hipMalloc(&d_particles, sizeof(Particle) * _num_particles);
      initParticles<<<grid, block>>>( d_particles, _num_particles );
      // ------------------------------------------


      if (_num_particles == 1)
      {
            _particles.push_back(Particle(0.1,
                                          vec3(1.0f,5.0f,1.0f),
                                          vec3(0.0f,0.0f,0.0f),
                                          vec3(0.0f,0.0f,0.0f)
                                          )
                                    );
      }
      if(form == PARTICLE_CUBE)
      {
            //Ramdomly generate positions of particles.
            int x_limit = 10;
            int z_limit = 10;
            glm::vec3 ranPos;
            for (int i = 0; i < _num_particles; i++)
            {
                  ranPos.x = -x_limit + static_cast <float> (rand()) / (static_cast <float> (RAND_MAX / (2 * x_limit)));
                  ranPos.z = -z_limit + static_cast <float> (rand()) / (static_cast <float> (RAND_MAX / (2 * z_limit)));
                  ranPos.y = 5 + static_cast <float> (rand()) / (static_cast <float> (RAND_MAX / (8)));
                  _particles.push_back(Particle(1,
                                                    ranPos,
                                                    vec3(0.0f,0.0f,0.0f),
                                                    vec3(0.0f,0.0f,0.0f)
                                                    )
                                              );
            }
      }
      if(form == PARTICLE_SPHERE)
      {
            //Ramdomly generate positions of particles.
            int radius = 2;
            glm::vec3 center = vec3(0,6,2);
            glm::vec3 ranPos;
            glm::vec3 dist;
            for (int i = 0; i < _num_particles; i++)
            {
                  ranPos = center;
                  do
                  {
                        dist.x = -radius + static_cast <float> (rand()) / (static_cast <float> (RAND_MAX / (2 * radius)));
                        dist.z = -radius + static_cast <float> (rand()) / (static_cast <float> (RAND_MAX / (2 * radius)));
                        dist.y = -radius + static_cast <float> (rand()) / (static_cast <float> (RAND_MAX / (2 * radius)));
                  }while (glm::length(dist) > radius);
                  ranPos += dist;

                  _particles.push_back(Particle(1,
                                                    ranPos,
                                                    vec3(0.0f,0.0f,0.0f),
                                                    vec3(0.0f,0.0f,0.0f)
                                                    )
                                              );
            }
      }
}

ParticleSystem::~ParticleSystem()
{
      hipFree(d_particles);
}


bool ParticleSystem::init(GLuint* programs)
{
      int i;
      for (i = 0; i < _num_particles; i++)
      {
            _positions.push_back(_particles[i].x);
            _colors.push_back(vec4(1.0f, 0.0f, 0.0f, 1.0f));
      }

	_model_matrix = mat4(1.0);

      glGenVertexArrays(1, &_vao);  //Create one vertex array object
	glBindVertexArray(_vao);
      glGenBuffers(2, _buffers); //Create two buffer objects, one for vertex positions and one for vertex colors

      glBindBuffer(GL_ARRAY_BUFFER, _buffers[0]);  //Buffers[0] wi ll be the position for each vertex
	glBufferData(GL_ARRAY_BUFFER, _num_particles * sizeof(vec3), NULL, GL_DYNAMIC_DRAW);
      //glBufferData(GL_ARRAY_BUFFER, _num_particles * sizeof(vec3), _positions.data(), GL_STATIC_DRAW);
	glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 0, 0);  //Do the shader plumbing here for this buffer
	glEnableVertexAttribArray(0);

	glBindBuffer(GL_ARRAY_BUFFER, _buffers[1]);  //Buffers[1] will be the color for each vertex
	glBufferData(GL_ARRAY_BUFFER, _num_particles * sizeof(vec4), _colors.data(), GL_STATIC_DRAW);
	glVertexAttribPointer(1, 4, GL_FLOAT, GL_FALSE, 0, 0);  //Do the shader plumbing here for this buffer
	glEnableVertexAttribArray(1);

      glLineWidth(1.0f);
      glPointSize(2.0f);

      _pvm_matrix_loc = glGetUniformLocation(programs[1], "_pvm_matrix");
      _projection_matrix_loc = glGetUniformLocation(programs[1], "_projection_matrix");
      _view_matrix_loc = glGetUniformLocation(programs[1], "_view_matrix");

	_initialized = true;
	return true;
}

void ParticleSystem::draw(GLuint* programs, mat4 proj_mat, mat4 view_mat)
{
      mat4 _pvm_matrix = proj_mat * view_mat * _model_matrix;
	glUniformMatrix4fv(_pvm_matrix_loc, 1, GL_FALSE, value_ptr(_pvm_matrix));
      glUniformMatrix4fv(_projection_matrix_loc, 1, GL_FALSE, value_ptr(proj_mat));
      glUniformMatrix4fv(_view_matrix_loc, 1, GL_FALSE, value_ptr(view_mat));

      glUseProgram(programs[1]);

      _positions.clear();

      int i;
      for (i = 0; i < _num_particles; i++)
      {
            //_positions.push_back(_particles[i].x);
      }

      glBindVertexArray(_vao);
	glBindBuffer(GL_ARRAY_BUFFER, _buffers[0]);  //Buffers[0] wi ll be the position for each vertex
      glBufferData(GL_ARRAY_BUFFER, _num_particles * sizeof(vec3), NULL, GL_DYNAMIC_DRAW);
	//glBufferData(GL_ARRAY_BUFFER, _num_particles * sizeof(vec3), _positions.data(), GL_STATIC_DRAW);
	glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 0, 0);  //Do the shader plumbing here for this buffer
	glEnableVertexAttribArray(0);


      // ---------- CUDA / OpenGL Data Mapping ----------

      // Register CUDA graphics resource with buffer, define what it will be used for.
      gpuErrchk( hipGraphicsGLRegisterBuffer( &res, _buffers[0], hipGraphicsRegisterFlagsWriteDiscard ) );

      // Map OpenGL Resource to CUDA device ptr
      gpuErrchk( hipGraphicsMapResources(1, &res) );
      gpuErrchk( hipGraphicsResourceGetMappedPointer(&device_ptr, &size, res) );

      // Kernel to move over the particle 5 in x dir
      cpyParticleDataToDraw<<<dim3(512), dim3(std::ceil(_num_particles / 512))>>>( d_particles, (vec3*)device_ptr , _num_particles );

      // Unmap OpenGL Resource from CUDA so the VBA can use it
      gpuErrchk( hipGraphicsUnmapResources(1, &res) );
      gpuErrchk( hipGraphicsUnregisterResource(res) );

      // ------------------------------------------------


      if (!_initialized)
      {
            cout << "ERROR : Cannot  render an object thats not initialized. ParticleSystem\n";
            return;
      }
      glDrawArrays(GL_POINTS, 0, _num_particles);
}
